#include "hip/hip_runtime.h"
// C++ program to find groups of unknown
// Points using K nearest neighbour algorithm.
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <fstream>
#include <sstream>
#include <hip/hip_runtime.h>
#define noOfClasses 2
struct Point
{
    int val;     // Co-ordinate of point
    float distance; // Distance from test point
};

int max(int freq[]){
    int m=freq[0];
    int index=0;
    for(int i=1;i<noOfClasses;i++){
        if(m<freq[i]){
            m=freq[i];
            index=i;
        }
    }
    return index+1;
}
// Used to sort an array of points by increasing
// order of distance
bool comparison(Point a, Point b)
{
    return (a.distance < b.distance);
}

int cpuClassify(int arr[], long n, int k,int attrib){
	int sum;
    long p=0;
    Point *result = (Point *)malloc(n*sizeof(Point));
    for (long i = 0; i < attrib*n; i=i+attrib)
        {
            sum=0;
            for(int j=0;j<attrib-1;j++){
                sum = sum + (arr[i+j]-arr[attrib*n+j])*(arr[i+j]-arr[attrib*n+j]);
            }
            result[p].distance=sqrt(sum);
            result[p].val=arr[i+attrib-1];

            p++;
        }
    // Sort the Points by distance from p
    std::sort(result, result+n, comparison);
    //for(long i=0;i<n;i++)
      //  printf("\n%f %d",result[i].distance,result[i].val);
    int freq[noOfClasses];
    for(int i=0;i<noOfClasses;i++)
        freq[i]=0;
    for (int i = 0; i < k; i++)
    {
        freq[result[i].val-1]++;
    }
    return max(freq);
}
__global__ void DistanceKernel(int *ga, int size, Point *gResult,int attrib)
{
	int i;
	int sum;
	i=(blockIdx.x*blockDim.x)+threadIdx.x;	
	if(i<size)
	{
		int z=i*attrib;
		sum=0;
		for(int j=0;j<attrib-1;j++){
                sum = sum + (ga[z+j]-ga[attrib*n+j])*(ga[z+j]-ga[attrib*n+j]);
            }	
		gResult[i].distance=sqrt(sum);
		gResult[i].val=ga[z+attrib-1];
	}
}

int gpuClassify(){
	Point *result = (Point *)malloc(n*sizeof(Point));
	
	int blockSize=128, blocks;
    hipError_t err;
    Point *ga;
    
    err=hipMalloc((void **)&ga,n*sizeof(Point));
    
             if (hipSuccess!=err)
            {
	                     printf("\n Memory allocation failed on GPU for ga");
	                     printf("\n error is- %s", hipGetErrorString(err));
	                     exit(EXIT_FAILURE);
			}
	    
	         if (hipSuccess!=hipMemcpy(ga,arr,n*sizeof(Point),hipMemcpyHostToDevice))
	             {
		                     printf("\n Error in copying ha to ga");
		                     exit(EXIT_FAILURE);
		     }
	
	err=hipMalloc((void **)&gResult,n*sizeof(Point));
    
            if (hipSuccess!=err)
            {
	                     printf("\n Memory allocation failed on GPU for result");
	                     printf("\n error is- %s", hipGetErrorString(err));
	                     exit(EXIT_FAILURE);
			}

	blocks=(int)(n/blockSize);
	if ((n%blockSize)>0)
		blocks++;
	printf("\n The number of blocks needed=%d", blocks);
	
	DistanceKernel<<<blocks,blockSize>>>(ga,n,gResult,attrib);     

	if (hipSuccess!=hipMemcpy(arr,ga,n*sizeof(int),hipMemcpyDeviceToHost))
		{
					printf("\n Error in copying ga to hb");
							exit(EXIT_FAILURE);
		} 
		
	if (hipSuccess!=hipMemcpy(result,gResult,n*sizeof(int),hipMemcpyDeviceToHost))
		{
					printf("\n Error in copying ga to hb");
							exit(EXIT_FAILURE);
		} 
  	
    // Sort the Points by distance from p
    std::sort(result, result+n, comparison);

    // Now consider the first k elements and only
    // two groups
    int freq[noOfClasses];
    for(int i=0;i<noOfClasses;i++)
        freq[i]=0;
    for (int i = 0; i < k; i++)
    {
        freq[result[i].val-1]++;
    }
    return max(freq);

}
void classifyAPoint(int arr[], int n, int k, int attrib)
{
    float timespentCPU, timespentGPU;
    clock_t start1, stop1;
    hipEvent_t start, stop; 
    hipEventCreate(&start); //Creates an event object 
    hipEventCreate(&stop);
		       
    //cpu time calculation
	start1=clock();
    int result =  cpuClassify(arr,n,k,attrib);      
    stop1=clock();
    timespentCPU = ((float)(stop1 - start1))/CLOCKS_PER_SEC;
    printf("\n result of cpuClassify is %d",result);
	printf("\n timespent on CPU=%f",timespentCPU);
    getchar();
    
    //cuda time calculation
	hipEventRecord(start, 0); //Timestamp, zero –default stream
	
	result=gpuClassify(arr,n,k,attrib);
	
	hipDeviceSynchronize();
	
	hipEventRecord(stop, 0); //Timestamp
	
	hipEventSynchronize(stop); 
	
	hipEventElapsedTime(&timespentGPU, start, stop); 
	
	printf("\n result of cpuClassify is %d",timespentGPU);
	printf("\n timespent on GPU=%f",timespentGPU);

	
}

// Driver code
int main()
{
    std::ifstream inFile;
    inFile.open("/home/student1/test.txt");
    if(!inFile)
      perror ( "Stream Failed to open because: " );
    int attrib=4;
    long n = 245057; // Number of data points
    int *arr = (int*)malloc((attrib*n+attrib)*sizeof(int));
    long i=0;

    std::string line;
    while (std::getline(inFile, line))
    {
        std::istringstream iss(line);
        //if (!(iss >> arr[i] >> arr[i] >> arr[i] >> arr[i] >> arr[i])) { break; }
        for(int j=0;j<attrib;j++){
                if (!(iss >> arr[i++])) { break; }
        }

        //printf("\n%f  %ld",arr[i].x,i);

    }

    /*Testing Point*/
    Point p;
    arr[i] = 7;
    arr[i+1] = 7;
    arr[i+2] = 0;

    // Parameter to decide groupr of the testing point
    int k = 1;
    classifyAPoint(arr, n, k,attrib);
    return 0;
}
